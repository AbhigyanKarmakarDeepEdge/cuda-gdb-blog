#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
__global__ void test__kernel(float* buf, float* result, size_t size)
{
	int i = (blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z) * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.z * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;	
	result[i] = buf[i] * buf[i+1];
}

void test__add()
{
	float* buf1, * sum;
	float* g_buf1, * g_sum;
	size_t size = 1000;

	buf1 = new float[size];
	sum = new float[size];

	hipMalloc((void**)&g_buf1, sizeof(float) * size);
	hipMalloc((void**)&g_sum, sizeof(float) * size);

	//Set vals for host buffers
	for (int i=0 ; i<size ; i++)	buf1[i] = i*0.1;

	hipMemcpy(g_buf1, buf1, sizeof(float) * (size), hipMemcpyHostToDevice);

	test__kernel << <dim3(1, size/32 + 1), dim3(32) >> > (g_buf1, g_sum, size);
	hipDeviceSynchronize();
	
	hipMemcpy(sum, g_sum, sizeof(float) * size, hipMemcpyDeviceToHost);
	for (int i=0 ; i<size ; i++)	std::cout << sum[i] << ", ";
	std::cout << std::endl;
}


int main()
{
	test__add();
}
