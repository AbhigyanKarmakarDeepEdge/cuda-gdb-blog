
#include <hip/hip_runtime.h>
constexpr size_t NUM_BLOCKS = 8;
constexpr size_t THREADS_PER_BLOCK = 16;

__global__ void example(int **data) {
	size_t threadID = threadIdx.x;
	size_t blockID = blockIdx.x * blockDim.x;
	size_t globalID = threadID + blockID;
	
	*(data[threadID]) = threadID;
	*(data[blockID]) = blockID;
	*(data[globalID]) = globalID;
}

int main(int argc, char *argv[]) {
	int *host_data[NUM_BLOCKS * THREADS_PER_BLOCK];
	int **dev_data;
	const int zero = 0;

	/*Allocate an integer for each thread in each block */
	for (int block = 0; block < NUM_BLOCKS; block++) 
	{
		for (int thread = 0; thread < THREADS_PER_BLOCK; thread++) 
		{
			int idx = thread + block * THREADS_PER_BLOCK;
			hipMalloc(&host_data[idx], sizeof(int));
			hipMemcpy(host_data[idx], &zero, sizeof(int),
			hipMemcpyHostToDevice);
		}
	}

	/* This inserts an error into block 4, thread 8*/
	host_data[4 * THREADS_PER_BLOCK + 8] = NULL;

	/* Copy the array of pointers to the device */
	hipMalloc((void**)&dev_data, sizeof(host_data));
	hipMemcpy(dev_data, host_data, sizeof(host_data), hipMemcpyHostToDevice);
	
	/* Execute example */
	example <<< NUM_BLOCKS, THREADS_PER_BLOCK >>> (dev_data);
	hipDeviceSynchronize();
}